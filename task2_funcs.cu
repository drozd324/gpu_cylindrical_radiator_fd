#include "hip/hip_runtime.h"
#include "task2_funcs.h"

__global__ void copySurface(hipSurfaceObject_t inputSurface, hipSurfaceObject_t outputSurface, int m, int n) {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx<n && idy<m) {
		float data;
		surf2Dread(&data, inputSurface, idx*4, idy);
		surf2Dwrite(data, outputSurface, idx*4, idy);
	}
}

__global__ void transformSurfaceToGlobal(hipSurfaceObject_t surface, float* gpu_data, int m, int n) {
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;

	if ( (idx < n) && (idy < m) ) {
		surf2Dread(&(gpu_data[idy*n+idx]), surface, idx*4 , idy);
    }
}

__global__ void transformGlobalToSurface(float* gpu_data, hipSurfaceObject_t surface, int m, int n) {
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;

	if ( (idx < n) && (idy < m) ) {
		surf2Dwrite(gpu_data[idy*n + idx], surface, idx*4, idy);
    }
}

__global__ void iterate_GPU_surface(hipSurfaceObject_t nextSurface, hipSurfaceObject_t prevSurface, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	float prevSurf_j_minus2; 
	float prevSurf_j_minus1;
	float prevSurf_j;
	float prevSurf_j_plus1;
	float prevSurf_j_plus2;

	float nextSurf_j;

	int mod_j_minus2;
	int mod_j_minus1;
	int mod_j_plus1;
	int mod_j_plus2;


	if ((idx<m) && (idy<n)){

		mod_j_minus2 = (idy - 2 + n) % n;
		mod_j_minus1 = (idy - 1 + n) % n;
		mod_j_plus1 = (idy + 1) % n;
		mod_j_plus2 = (idy + 2) % n;

		// read
		surf2Dread(&prevSurf_j_minus2, prevSurface, idx*4, mod_j_minus2);
		surf2Dread(&prevSurf_j_minus1, prevSurface, idx*4, mod_j_minus1);
		surf2Dread(&prevSurf_j       , prevSurface, idx*4, idy         );
		surf2Dread(&prevSurf_j_plus1 , prevSurface, idx*4, mod_j_plus1 );
		surf2Dread(&prevSurf_j_plus2 , prevSurface, idx*4, mod_j_plus2 );

		// compute
		nextSurf_j = ((1.60*prevSurf_j_minus2) + (1.55*prevSurf_j_minus1) + prevSurf_j + (0.60*prevSurf_j_plus1) + (0.25*prevSurf_j_plus2)) / ((float)(5.0));
		
		// write
		surf2Dwrite(nextSurf_j, nextSurface, idx*4, idy);
	}
}

__global__ void calculate_avg_temp_GPU(float* matrix, int m, int n, float* thermometer){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if ((idx<m) && (idy<n)){
		atomicAdd(&(thermometer[idx]), matrix[idx*n + idy]);
	}

	//__syncthreads();

	if ((idx<m) && (idy == 0)){
		thermometer[idx] /= n;
	}
}


__global__ void iterate_GPU_global(float* nextMatrix, float* previousMatrix, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int mod_j_minus2;
	int mod_j_minus1;
	int mod_j_plus1;
	int mod_j_plus2;

	if ((idx<m) && (idy<n)){
		mod_j_minus2 = (idy - 2 + n) % n;
		mod_j_minus1 = (idy - 1 + n) % n;
		mod_j_plus1 = (idy + 1) % n;
		mod_j_plus2 = (idy + 2) % n;


		nextMatrix[idx*n + idy] = ((1.60*previousMatrix[idx*n + mod_j_minus2]) + 
									(1.55*previousMatrix[idx*n + mod_j_minus1]) + 
									previousMatrix[idx*n + idy] + 
									(0.60*previousMatrix[idx*n + mod_j_plus1]) +
									(0.25*previousMatrix[idx*n + mod_j_plus2]));
		nextMatrix[idy*n + idy] /= (float)(5.0);
	}
}

__global__ void init_matrix_GPU_global(float* matrix, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if ((idx<m) && (idy==0)){
		matrix[idx*n + 0] = 0.98 * (float)((idx+1)*(idx+1)) / (float)(n*n); 
	}
	
	if ((idx<m) &&  ((1<=idy) && (idy<n))){
		matrix[idx*n + idy] = ( 0.98 * (float)((idx+1)*(idx+1)) / (float)(n*n) )
								* ( ((float)((m-idy)*(m-idy))) / ((float)(m*m)));	
	}
}

// wont work
//__global__ void iterate_GPU_texture(hipTextureObject_t nextTexture, hipTextureObject_t prevTexture, int m, int n){
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	int idy = blockIdx.y * blockDim.y + threadIdx.y;
//
//	if ((idx<n) && (idy<m)){
//		// compute
//		tex2D<textureType>(nextTexture, idx, idy) = ( (1.60*tex2D<textureType>(prevTexture, idx, idy-2)  + 
//														(1.55*tex2D<textureType>(prevTexture, idx, idy-1)) +
//														tex2D<textureType>(prevTexture, idx, idy)          +
//														(0.60*tex2D<textureType>(prevTexture, idx, idy+1)) +
//														(0.25*tex2D<textureType>(prevTexture, idx, idy+2)) ) 
//														/ ((float)(5.0));
//	}
//}
