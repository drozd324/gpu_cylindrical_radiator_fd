#include "hip/hip_runtime.h"
#include "task2_funcs.h"

//__global__ void copySurface(hipSurfaceObject_t inputSurface, hipSurfaceObject_t outputSurface, int m, int n) {
//	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
//
//	if (idx<n && idy<m) {
//		float data;
//		surf2Dread(&data, inputSurface, idx*4, idy);
//		surf2Dwrite(data, outputSurface, idx*4, idy);
//	}
//}

__global__ void transformSurfaceToGlobal(hipSurfaceObject_t surface, float* gpu_data, int m, int n) {
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;

	if ((idx<m) && (idy<n)) {
		surf2Dread(&(gpu_data[idx*n + idy]), surface, idx*4 , idy);
    }
}

__global__ void transformGlobalToSurface(float* gpu_data, hipSurfaceObject_t surface, int m, int n) {
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;

	if ((idx<m) && (idy<n)) {
		surf2Dwrite(gpu_data[idx*n + idy], surface, idx*4, idy);
    }
}

__global__ void iterate_GPU_surface(hipSurfaceObject_t nextSurface, hipSurfaceObject_t prevSurface, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	float prevSurf_j_minus2; 
	float prevSurf_j_minus1;
	float prevSurf_j;
	float prevSurf_j_plus1;
	float prevSurf_j_plus2;

	float nextSurf_j;

	int mod_j_minus2;
	int mod_j_minus1;
	int mod_j_plus1;
	int mod_j_plus2;


	if ((idx<m) && (idy<n)){

		mod_j_minus2 = (idy - 2 + n) % n;
		mod_j_minus1 = (idy - 1 + n) % n;
		mod_j_plus1 = (idy + 1) % n;
		mod_j_plus2 = (idy + 2) % n;

		// read
		surf2Dread(&prevSurf_j_minus2, prevSurface, idx*4, mod_j_minus2);
		surf2Dread(&prevSurf_j_minus1, prevSurface, idx*4, mod_j_minus1);
		surf2Dread(&prevSurf_j       , prevSurface, idx*4, idy         );
		surf2Dread(&prevSurf_j_plus1 , prevSurface, idx*4, mod_j_plus1 );
		surf2Dread(&prevSurf_j_plus2 , prevSurface, idx*4, mod_j_plus2 );

		// compute
		nextSurf_j = ((1.60*prevSurf_j_minus2) + (1.55*prevSurf_j_minus1) + prevSurf_j + (0.60*prevSurf_j_plus1) + (0.25*prevSurf_j_plus2)) / ((float)(5.0));
		
		// write
		surf2Dwrite(nextSurf_j, nextSurface, idx*4, idy);
	}
}

__global__ void calculate_avg_temp_GPU(float* matrix, int m, int n, float* thermometer){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if ((idx<m) && (idy<n)){
		atomicAdd(&(thermometer[idx]), matrix[idx*n + idy]);
	}

	//__syncthreads();

	if ((idx<m) && (idy == 0)){
		thermometer[idx] /= n;
	}
}


// iteration function for global memory data
__global__ void iterate_GPU_global(float* nextMatrix, float* previousMatrix, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int mod_j_minus2;
	int mod_j_minus1;
	int mod_j_plus1;
	int mod_j_plus2;

	if ((idx<m) && (idy<n)){
		mod_j_minus2 = (idy - 2 + n) % n;
		mod_j_minus1 = (idy - 1 + n) % n;
		mod_j_plus1 = (idy + 1) % n;
		mod_j_plus2 = (idy + 2) % n;


		nextMatrix[idx*n + idy] = ((1.60*previousMatrix[idx*n + mod_j_minus2]) + 
									(1.55*previousMatrix[idx*n + mod_j_minus1]) + 
									previousMatrix[idx*n + idy] + 
									(0.60*previousMatrix[idx*n + mod_j_plus1]) +
									(0.25*previousMatrix[idx*n + mod_j_plus2]));
		nextMatrix[idy*n + idy] /= (float)(5.0);
	}
}


// hw1 reduce code

__global__ void sum_rows_gpu(int m, int n, float* a, float* v){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < m){
        v[idx] = 0;
       	for (int j=0; j<n; j++){ 
           	v[idx] += (0 < (a[idx*n + j])) ? a[idx*n + j] : -a[idx*n + j];
       	}
	}
}

// hw1 reduce code end
