#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include "task1_funcs.h"
#include "task2_funcs.h"

int main(int argc, char *argv[]) {

	int option;
	int m = 32;
	int n = 32;
	int iter = 10;
	int calc_cpu = 0;
	int calc_avg_temp = 0;
	int calc_avg_temp_no_atomic = 0;
	int show_timings_next_to_eachother = 0;
	int save_data = 0;

	char filename[100];
	sprintf(filename, "writeup/task3.csv");

	int block_size_x = 2; //threads per block
	int block_size_y = 2; //threads per block

    while ((option = getopt(argc, argv, "m:n:p:x:y:aActs")) != -1) {
        switch (option) {
            case 'm': // set num cols m of matrix
	            m = atoi(optarg);
				break;
			case 'n': // set num rows n of matrix
				n = atoi(optarg);
				break;
			case 'p': // set iterations
				iter = atoi(optarg);
				break;
			case 'x': // set block_size_x
				block_size_x = atoi(optarg);
				break;
			case 'y': // set block_size_y
				block_size_y = atoi(optarg);
				break;
			case 'a': // sets caclulation of average temperature for each row
				calc_avg_temp = 1;
				break;
			case 'A': // sets caclulation of average temperature for each row with hw1 implementation
				calc_avg_temp = 1;
				calc_avg_temp_no_atomic = 1;
				sprintf(filename, "writeup/task3_old_reduce.csv");
				
				break;
			case 'c': // caclulates cpu version of algorithm
				calc_cpu = 1;
				break;
			case 't': // caclulates all timings
				show_timings_next_to_eachother = 1;
				calc_cpu = 1;
				calc_avg_temp = 1;
				break;
			case 's': // saves all the data to a csv file
				show_timings_next_to_eachother = 1;
				calc_cpu = 1;
				calc_avg_temp = 1;
				save_data = 1;
				break;
        }
    }

	// GPU Calculation //
	//=================================================================//
	printf("\n//======================================//\n");
	printf("              GPU Calculation               \n");
	printf("//======================================//\n\n");

	if ( ((n*m) % (block_size_x*block_size_y)) != 0){
		fprintf(stderr, "[ERROR]: block size (number of threads per block) doesnt divide the total size of the matrix\n");
		return 1;
	}

	int N = n;
	int M = m;
	dim3 dimBlock (block_size_x, block_size_y);
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1),(M/dimBlock.y) + (!(M%dimBlock.y)?0:1) );
		
	//cuda timings
	hipEvent_t start, finish;
	hipEventCreate(&start);
	hipEventCreate(&finish);
	float elapsedTime;

	double time_allocating;
	//double time_transfering_to_gpu;
	double time_compute;
	double time_calc_averages;
	//double time_transfering_to_cpu;

	// allocalte matrices a_h, b_h on host
	double* a_h;
	double* b_h;
	a_h = (double*) malloc(m*n * sizeof(double));
	b_h = (double*) malloc(m*n * sizeof(double));

	// init on cpu
	init_matrix(a_h, m, n);
	init_matrix(b_h, m, n);

//	printf("before \n");
//	printf("printing a_h\n");
//	print_matrix(a_h, m, n);
	
	// alloc on device global memory
	double* a_d;
	double* b_d;
	hipEventRecord(start, 0);
	hipMalloc((void**)&a_d, m*n * sizeof(double));
	hipMalloc((void**)&b_d, m*n * sizeof(double));
	hipEventRecord(finish, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time allocating on GPU = %lf\n", elapsedTime);
	time_allocating = elapsedTime;

	// alloc surface memory
	int width = n;
	int height = m;
	int size = width * height * sizeof(double);
	double* host_input_data = (double*)malloc(size);

	//////////////////////////////TEXTURE/SURFACE MEMORY SHTUFF////////////////////////////////////////////////

	// Allocate CUDA arrays in device memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(64, 64, 0, 0, hipChannelFormatKindFloat);

	hipArray* array_a_d; // array a on device
	hipArray* array_b_d;
	hipMallocArray(&array_a_d, &channelDesc, width, height, hipArraySurfaceLoadStore);
	hipMallocArray(&array_b_d, &channelDesc, width, height, hipArraySurfaceLoadStore);

	// Copy to device memory some data located at address host_input_data  in host memory
	const size_t spitch = width * sizeof(double);
	hipMemcpy2DToArray(array_a_d, 0, 0, a_h, spitch, width * sizeof(double), height, hipMemcpyHostToDevice);
	hipMemcpy2DToArray(array_b_d, 0, 0, b_h, spitch, width * sizeof(double), height, hipMemcpyHostToDevice);

	// Create the surface objects
	// Declare the surface memory arrays
	hipSurfaceObject_t aSurf = 0;
	hipSurfaceObject_t bSurf = 0;

	// Set up the structure for the surfaces
	struct hipResourceDesc resDesc_aSurf;
	memset(&resDesc_aSurf, 0, sizeof(resDesc_aSurf));
	resDesc_aSurf.resType = hipResourceTypeArray;
	resDesc_aSurf.res.array.array = array_a_d;

	struct hipResourceDesc resDesc_bSurf;
	memset(&resDesc_bSurf, 0, sizeof(resDesc_bSurf));
	resDesc_bSurf.resType = hipResourceTypeArray;
	resDesc_bSurf.res.array.array = array_b_d;

	// Bind the arrays to the surface objects
	hipCreateSurfaceObject(&aSurf, &resDesc_aSurf);
	hipCreateSurfaceObject(&bSurf, &resDesc_bSurf);

	//////////////////////////////////////////////////////////////////////////////////////////////////	


	// copy to gpu
	hipEventRecord(start, 0);
	hipMemcpy(a_d, a_h, m*n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, m*n * sizeof(double), hipMemcpyHostToDevice);
	// copy into surface memory
	transformGlobalToSurface<<<dimGrid, dimBlock>>>(a_d, aSurf, m, n); 
	transformGlobalToSurface<<<dimGrid, dimBlock>>>(b_d, bSurf, m, n); 
	hipEventRecord(finish, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time transfering to GPU = %lf\n", elapsedTime);
	//time_transfering_to_gpu = elapsedTime;

	hipEventRecord(start, 0);
	for (int i=0; i<iter; i++){
		iterate_GPU_surface<<<dimGrid, dimBlock>>>(aSurf, bSurf, m, n);
		iterate_GPU_surface<<<dimGrid, dimBlock>>>(bSurf, aSurf, m, n);
	}
	hipEventRecord(finish, 0);
		
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time for compute on GPU = %lf\n", elapsedTime);
	time_compute = elapsedTime;

	// copy into RAM
	hipEventRecord(start, 0);
	transformSurfaceToGlobal<<<dimGrid, dimBlock>>>(aSurf, a_d, m, n); 
	transformSurfaceToGlobal<<<dimGrid, dimBlock>>>(bSurf, b_d, m, n); 
	hipMemcpy(a_h, a_d, m*n * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(b_h, b_d, m*n * sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(finish, 0);
	
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time to transfer to RAM = %lf\n", elapsedTime);	
	//time_transfering_to_cpu = elapsedTime;

	double* thermometer_d;
	double* thermometer_h = (double*) malloc(m * sizeof(double));
	if (calc_avg_temp == 1){
		hipMalloc((void**)&thermometer_d, m * sizeof(double));
		hipMemset(thermometer_d, 0, m * sizeof(double));

		if (calc_avg_temp_no_atomic == 0){
			hipEventRecord(start, 0);
			calculate_avg_temp_GPU<<<dimGrid, dimBlock>>>(a_d, m, n, thermometer_d);
			hipEventRecord(finish, 0);
		} else { // from hw1
			hipEventRecord(start, 0);
 			sum_rows_gpu<<<dimGrid, dimBlock>>>(m, n, a_d, thermometer_d);
			hipEventRecord(finish, 0);

		}

		hipEventSynchronize(start);
		hipEventSynchronize(finish);
		hipEventElapsedTime(&elapsedTime, start, finish);
		printf("Time to calculate averages on GPU = %lf\n", elapsedTime);	
		time_calc_averages = elapsedTime;

	}	
	hipMemcpy(thermometer_h, thermometer_d, m * sizeof(double), hipMemcpyDeviceToHost);

	

	// end //
	//=================================================================//
	
		
	clock_t time_start;			
	clock_t time_end;			
	double cpu_time_allocating;
	double cpu_time_compute;
	double cpu_time_calc_averages;

	double* a;
	double* b;
	double* thermometer;

	// CPU Calculation //
	//=================================================================//
	if (calc_cpu == 1){
			printf("\n//======================================//\n");
			printf("               CPU Calculation              \n");
			printf("//======================================//\n\n");

	
			// allocalte matrices a, b
			time_start = clock();
			a = (double*) malloc(m*n * sizeof(double));
			b = (double*) malloc(m*n * sizeof(double));
			time_end = clock();
			cpu_time_allocating = (double)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3);
			printf("Time allocating on CPU = %lf\n", cpu_time_allocating);
			
			time_start = clock();
			init_matrix(a, m, n);
			init_matrix(b, m, n);
			time_end = clock();
			printf("Time initialising matrices on CPU = %lf\n", (double)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3));
			
//			printf("before \n");
//			printf("printing a\n");
//			print_matrix(a, m, n);

			time_start = clock();
			for (int i=0; i<iter; i++){
				iterate(a, b, m, n);
				iterate(b, a, m, n);
			}
			time_end = clock();
			cpu_time_compute = (double)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3);
			printf("Time for compute on CPU = %lf\n", cpu_time_compute);
				
			thermometer = (double*) calloc(m, sizeof(double));
			if (calc_avg_temp == 1){

				time_start = clock();
				calculate_avg_temp(a, m, n, thermometer);
				time_end = clock();
				cpu_time_calc_averages = (double)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3);
				printf("Time to calculate averages on CPU = %lf\n", cpu_time_calc_averages);

			}	
	}
	// end //
	//=================================================================//

		

	if (show_timings_next_to_eachother == 1){
		// compute errors
		printf("\n");
		double max_matrix_diff = max_diff(a_h, a, m, n);
		double max_avg_diff = max_diff(thermometer_h, thermometer, m, 1);

		printf("\n//======================================//\n");
		printf("      SHOWING MAIN TIMINGS AND SPEEDUPS     \n");
		printf("//======================================//\n\n");
			
		printf("Allocating memory    | CPU: %lf | GPU: %lf | Speedup: %lf\n", 
				cpu_time_allocating, time_allocating, cpu_time_allocating/time_allocating);
		printf("Main compute         | CPU: %lf | GPU: %lf | Speedup: %lf\n", 
				cpu_time_compute, time_compute, cpu_time_compute/time_compute);
		printf("Calculating averages | CPU: %lf | GPU: %lf | Speedup: %lf\n", 
				cpu_time_calc_averages, time_calc_averages, cpu_time_calc_averages/time_calc_averages);
		printf("\n");

		printf("Maximum difference between CPU and GPU | Avg temp: %lf\n                                       | Matrices: %lf\n",max_avg_diff, max_matrix_diff); 
		
		printf("\n");

		if (save_data == 1){
			FILE *fp = fopen(filename, "a");

			fprintf(fp, "%d,%d,%d,%d,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf", 
					m, n, block_size_x, block_size_y,
					cpu_time_allocating, time_allocating, cpu_time_allocating/time_allocating,
					cpu_time_compute, time_compute, cpu_time_compute/time_compute,
					cpu_time_calc_averages, time_calc_averages, cpu_time_calc_averages/time_calc_averages);
			fprintf(fp, "\n");
			fclose(fp);
		}

//		printf("after \n");
//		printf("printing a_h\n");
//		print_matrix(a_h, m, n);
//		printf("printing a\n");
//		print_matrix(a, m, n);

	}

	// free cuda parts
	free(a_h);
	free(b_h);
	free(thermometer_h);
	hipFree(a_d);
	hipFree(b_d);
	hipFreeArray(array_a_d); // frees cuda surface
	hipFreeArray(array_b_d);
	hipFree(thermometer_d);

	// free cpu parts
	if (calc_cpu == 1){
		free(a);
		free(b);
		free(thermometer);
	}


    return 0;
}
