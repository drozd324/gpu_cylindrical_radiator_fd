#include "hip/hip_runtime.h"
#include "task2_funcs.h"

__global__ void copySurface(hipSurfaceObject_t inputSurface, hipSurfaceObject_t outputSurface, int width, int height) {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx<width && idy<height) {
		float data;
		surf2Dread(&data, inputSurface, idx * 4, idy);
		surf2Dwrite(data, outputSurface, idx * 4, idy);
	}
}

__global__ void transformSurfaceToGlobal(hipSurfaceObject_t surface, float* gpu_data, int width, int height) {
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;

	if ( (idx < width) && (idy < height) ) {
		surf2Dread(&(gpu_data[idy*width+idx]), surface, idx*4 , idy);
    }
}

__global__ void transformGlobalToSurface(float* gpu_data, hipSurfaceObject_t surface, int width, int height) {
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;

	if ( (idx < width) && (idy < height) ) {
		surf2Dwrite(gpu_data[idy*width + idx], surface, idx * 4, idy);
    }
}

__global__ void iterate_GPU(hipSurfaceObject_t nextSurface, hipSurfaceObject_t prevSurface, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	float prevSurf_j_minus2; 
	float prevSurf_j_minus1;
	float prevSurf_j;
	float prevSurf_j_plus1;
	float prevSurf_j_plus2;

	float nextSurf_j;

	int mod_j_minus2;
	int mod_j_minus1;
	int mod_j_plus1;
	int mod_j_plus2;


	if (idx<m){
		if (idy<n){

			mod_j_minus2 = (idy - 2 + m) % m;
			mod_j_minus1 = (idy - 1 + m) % m;
			mod_j_plus1 = (idy + 1) % m;
			mod_j_plus2 = (idy + 2) % m;

			// read
			surf2Dread(&prevSurf_j_minus2, prevSurface, idx*4, mod_j_minus2);
			surf2Dread(&prevSurf_j_minus1, prevSurface, idx*4, mod_j_minus1);
			surf2Dread(&prevSurf_j       , prevSurface, idx*4, idy         );
			surf2Dread(&prevSurf_j_plus1 , prevSurface, idx*4, mod_j_plus1 );
			surf2Dread(&prevSurf_j_plus2 , prevSurface, idx*4, mod_j_plus2 );

			// compute
			nextSurf_j = ((1.60*prevSurf_j_minus2) + (1.55*prevSurf_j_minus1) + prevSurf_j + (0.60*prevSurf_j_plus1) + (0.25*prevSurf_j_plus2)) / ((float)(5.0));
			
			// write
			surf2Dwrite(nextSurf_j, nextSurface, idx * 4, idy);
		}
	}
}

__global__ void calculate_avg_temp_GPU(float* matrix, int m, int n, float* thermometer){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;


	if (idx<m){
		if (idy<n){
			atomicAdd(&(thermometer[idx]), matrix[idx*n + idy]);
		}
	}

	//__syncthreads();

	if (idx<m){
		if (idy == 0){
			thermometer[idx] /= m;
		}
	}
}

