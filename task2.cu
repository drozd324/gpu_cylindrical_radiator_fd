#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include "task1_funcs.h"
#include "task2_funcs.h"

int main(int argc, char *argv[]) {

	int option;
	int m = 32;
	int n = 32;
	int iter = 10;
	int calc_cpu = 0;
	int calc_avg_temp = 0;
	int show_timings_next_to_eachother = 0;

    while ((option = getopt(argc, argv, "m:n:p:act")) != -1) {
        switch (option) {
            case 'm': // set num cols m of matrix
	            m = atoi(optarg);
				break;
			case 'n': // set num rows n of matrix
				n = atoi(optarg);
				break;
			case 'p': // set iterations
				iter = atoi(optarg);
				break;
			case 'a': // sets caclulation of average temperature for each row
				calc_avg_temp = 1;
				break;
			case 'c': // caclulates cpu version of algorithm
				calc_cpu = 1;
				break;
			case 't': // caclulates cpu version of algorithm
				show_timings_next_to_eachother = 1;
				calc_avg_temp = 1;
				break;
        }
    }

	
	// GPU Calculation //
	//=================================================================//

	int N = n;
	int M = m;
	int block_size = 4;
	dim3 dimBlock (block_size, block_size);
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1),(M/dimBlock.y) + (!(M%dimBlock.y)?0:1) );
		
	if (( (n*m) % block_size) != 0){
		fprintf(stderr, "ERROR: block size (number of threads per block) doesnt divide the total size of the matrix\n");
		return 1;
	}

	//cuda timings
	hipEvent_t start, finish;
	hipEventCreate(&start);
	hipEventCreate(&finish);
	float elapsedTime;

	float time_allocating;
	float time_transfering_to_gpu;
	float time_compute_gpu;
	float time_calc_averages;
	float time_transfering_to_cpu;

	// allocalte matrices a_h, b_h on host
	float* a_h;
	float* b_h;
	a_h = (float*) malloc(m*n * sizeof(float));
	b_h = (float*) malloc(m*n * sizeof(float));
	
	// alloc on device
	hipEventRecord(start, 0);
	float* a_d;
	float* b_d;
	hipMalloc((void**)&a_d, m*n * sizeof(float));
	hipMalloc((void**)&b_d, m*n * sizeof(float));
	hipEventRecord(finish, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time allocating on GPU = %f\n", elapsedTime);
	time_allocating = elapsedTime;

	hipEventRecord(start, 0);
	init_matrix_GPU<<<dimGrid, dimBlock>>>(a_d, m, n);
	init_matrix_GPU<<<dimGrid, dimBlock>>>(b_d, m, n);
	hipEventRecord(finish, 0);
	
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time initialising matrices on GPU = %f\n", elapsedTime);

	hipEventRecord(start, 0);
	for (int i=0; i<iter; i++){
		iterate_GPU<<<dimGrid, dimBlock>>>(a_d, b_d, m, n);
		iterate_GPU<<<dimGrid, dimBlock>>>(b_d, a_d, m, n);
	}
	hipEventRecord(finish, 0);
		
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time for compute on GPU = %f\n", elapsedTime);
	time_compute_gpu = elapsedTime;

	if (calc_avg_temp == 1){
		float* thermometer_d;
		hipMalloc((void**)&thermometer_d, m * sizeof(float));

		hipEventRecord(start, 0);
		calculate_avg_temp_GPU<<<dimGrid, dimBlock>>>(a_d, m, n, thermometer_d);
		hipEventRecord(finish, 0);

		hipEventSynchronize(start);
		hipEventSynchronize(finish);
		hipEventElapsedTime(&elapsedTime, start, finish);
		printf("Time to calculate averages on GPU = %f\n", elapsedTime);	
		time_calc_averages = elapsedTime;

		hipFree(thermometer_d);
	}	


	hipEventRecord(start, 0);
	hipMemcpy(a_h, a_d, m*n * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(b_h, b_d, m*n * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(finish, 0);
	
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time to transfer to RAM = %f\n", elapsedTime);	
	time_transfering_to_cpu = elapsedTime;
	
	// free
	free(a_h);	
	free(b_h);	
	hipFree(a_d);	
	hipFree(b_d);	

	// end //
	//=================================================================//

	if (calc_cpu == 1){
			// allocalte matrices a, b

			float time_start, time_end;			
			float time_allocating;
			float time_transfering_to_gpu;
			float time_compute_gpu;
			float time_calc_averages;
			float time_transfering_to_cpu;
	
			time_start = walltime();
			// allocalte matrices a_h, b_h on host
			float* a;
			float* b;
			a = (float*) malloc(m*n * sizeof(float));
			b = (float*) malloc(m*n * sizeof(float));
			time_end = walltime();
			
			init_matrix(a, m, n);
			init_matrix(b, m, n);
			
			for (int i=0; i<iter; i++){
				iterate(a, b, m, n);
				iterate(b, a, m, n);
			}
				
			if (calc_avg_temp == 1){
				float* thermometer = (float*) calloc(m, sizeof(float));
				calculate_avg_temp(a, m, n, thermometer);
				free(thermometer);
			}	
			
			// free matrices a, b
			free(a);
			free(b);
	}

    return 0;
}
