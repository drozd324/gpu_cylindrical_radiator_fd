#include "hip/hip_runtime.h"
#include "task2_funcs.h"

__global__ void init_matrix_GPU(float* matrix, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx<m){
		if (idy==0){
			matrix[idx*n + 0] = 0.98 * (float)((idx+1)*(idx+1)) / (float)(n*n); 
		}
	}
	
	if (idx<m){
		if ((1<=idy) && (idy<n)){
			matrix[idx*n + idy] = ( 0.98 * (float)((idx+1)*(idx+1)) / (float)(n*n) )
									* ( ((float)((m-idy)*(m-idy))) / ((float)(m*m)));	
		}
	}
}

//__global__ void iterate_GPU(float* nextMatrix, float* previousMatrix, int m, int n){
__global__ void iterate_GPU(float* nextMatrix, float* previousMatrix, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx<m){
		if ((2<=idy) && (idy<n-2)){
			nextMatrix[idx*n + idy] = ((1.60*previousMatrix[idx*n + idy-2]) + 
									(1.55*previousMatrix[idx*n + idy-1]) + 
									previousMatrix[idx*n + idy] + 
									(0.60*previousMatrix[idx*n + idy+1]) +
									(0.25*previousMatrix[idx*n + idy+2]));
			nextMatrix[idx*n + idy] /= (float)(5.0);
		}
	}
}



__global__ void iterate_GPU_old(float* nextMatrix, float* previousMatrix, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	//__syncthreads();

	if (idx<m){
		if ((2<=idy) && (idy<n-2)){
			nextMatrix[idx*n + idy] = ((1.60*previousMatrix[idx*n + idy-2]) + 
									(1.55*previousMatrix[idx*n + idy-1]) + 
									previousMatrix[idx*n + idy] + 
									(0.60*previousMatrix[idx*n + idy+1]) +
									(0.25*previousMatrix[idx*n + idy+2]));
			nextMatrix[idx*n + idy] /= (float)(5.0);
		}
	}


__global__ void calculate_avg_temp_GPU(float* matrix, int m, int n, float* thermometer){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;


	if (idx<m){
		if (idy<n){
			atomicAdd(&(thermometer[idx]), matrix[idx*n + idy]);
		}
	}

	__syncthreads();

	if (idx<m){
		if (idy == 0){
			thermometer[idx] /= m;
		}
	}
}

