#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include "task1_funcs.h"
#include "task2_funcs.h"

int main(int argc, char *argv[]) {

	int option;
	int m = 32;
	int n = 32;
	int iter = 10;
	int calc_cpu = 0;
	int calc_avg_temp = 0;
	int calc_avg_temp_no_atomic = 0;
	int show_timings_next_to_eachother = 0;
	int save_data = 0;

	char filename[100];
	sprintf(filename, "writeup/task3.csv");

	int block_size_x = 2; //threads per block
	int block_size_y = 2; //threads per block

    while ((option = getopt(argc, argv, "m:n:p:x:y:aActs")) != -1) {
        switch (option) {
            case 'm': // set num cols m of matrix
	            m = atoi(optarg);
				break;
			case 'n': // set num rows n of matrix
				n = atoi(optarg);
				break;
			case 'p': // set iterations
				iter = atoi(optarg);
				break;
			case 'x': // set block_size_x
				block_size_x = atoi(optarg);
				break;
			case 'y': // set block_size_y
				block_size_y = atoi(optarg);
				break;
			case 'a': // sets caclulation of average temperature for each row
				calc_avg_temp = 1;
				break;
			case 'A': // sets caclulation of average temperature for each row with hw1 implementation
				calc_avg_temp = 1;
				calc_avg_temp_no_atomic = 1;
				sprintf(filename, "writeup/task3_old_reduce.csv");
				
				break;
			case 'c': // caclulates cpu version of algorithm
				calc_cpu = 1;
				break;
			case 't': // caclulates all timings
				show_timings_next_to_eachother = 1;
				calc_cpu = 1;
				calc_avg_temp = 1;
				break;
			case 's': // saves all the data to a csv file
				show_timings_next_to_eachother = 1;
				calc_cpu = 1;
				calc_avg_temp = 1;
				save_data = 1;
				break;
        }
    }

	// GPU Calculation //
	//=================================================================//
	printf("\n//======================================//\n");
	printf("              GPU Calculation               \n");
	printf("//======================================//\n\n");

	if ( ((n*m) % (block_size_x*block_size_y)) != 0){
		fprintf(stderr, "[ERROR]: block size (number of threads per block) doesnt divide the total size of the matrix\n");
		return 1;
	}

	int N = n;
	int M = m;
	dim3 dimBlock (block_size_x, block_size_y);
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1),(M/dimBlock.y) + (!(M%dimBlock.y)?0:1) );
		
	//cuda timings
	hipEvent_t start, finish;
	hipEventCreate(&start);
	hipEventCreate(&finish);
	float elapsedTime;

	float time_allocating;
	//float time_transfering_to_gpu;
	float time_compute;
	float time_calc_averages;
	//float time_transfering_to_cpu;

	// allocalte matrices a_h, b_h on host
	float* a_h;
	float* b_h;
	a_h = (float*) malloc(m*n * sizeof(float));
	b_h = (float*) malloc(m*n * sizeof(float));

	// init on cpu
	init_matrix(a_h, m, n);
	init_matrix(b_h, m, n);

//	printf("before \n");
//	printf("printing a_h\n");
//	print_matrix(a_h, m, n);
	
	// alloc on device global memory
	float* a_d;
	float* b_d;
	hipEventRecord(start, 0);
	hipMalloc((void**)&a_d, m*n * sizeof(float));
	hipMalloc((void**)&b_d, m*n * sizeof(float));
	hipEventRecord(finish, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time allocating on GPU = %.17f\n", elapsedTime);
	time_allocating = elapsedTime;

	// alloc surface memory
	int width = n;
	int height = m;
	int size = width * height * sizeof(float);
	float* host_input_data = (float*)malloc(size);

	//////////////////////////////TEXTURE/SURFACE MEMORY SHTUFF////////////////////////////////////////////////

	// Allocate CUDA arrays in device memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);

	hipArray* array_a_d; // array a on device
	hipArray* array_b_d;
	hipMallocArray(&array_a_d, &channelDesc, width, height, hipArraySurfaceLoadStore);
	hipMallocArray(&array_b_d, &channelDesc, width, height, hipArraySurfaceLoadStore);

	// Copy to device memory some data located at address host_input_data  in host memory
	const size_t spitch = width * sizeof(float);
	hipMemcpy2DToArray(array_a_d, 0, 0, a_h, spitch, width * sizeof(float), height, hipMemcpyHostToDevice);
	hipMemcpy2DToArray(array_b_d, 0, 0, b_h, spitch, width * sizeof(float), height, hipMemcpyHostToDevice);

	// Create the surface objects
	// Declare the surface memory arrays
	hipSurfaceObject_t aSurf = 0;
	hipSurfaceObject_t bSurf = 0;

	// Set up the structure for the surfaces
	struct hipResourceDesc resDesc_aSurf;
	memset(&resDesc_aSurf, 0, sizeof(resDesc_aSurf));
	resDesc_aSurf.resType = hipResourceTypeArray;
	resDesc_aSurf.res.array.array = array_a_d;

	struct hipResourceDesc resDesc_bSurf;
	memset(&resDesc_bSurf, 0, sizeof(resDesc_bSurf));
	resDesc_bSurf.resType = hipResourceTypeArray;
	resDesc_bSurf.res.array.array = array_b_d;

	// Bind the arrays to the surface objects
	hipCreateSurfaceObject(&aSurf, &resDesc_aSurf);
	hipCreateSurfaceObject(&bSurf, &resDesc_bSurf);

	//////////////////////////////////////////////////////////////////////////////////////////////////	


	// copy to gpu
	hipEventRecord(start, 0);
	hipMemcpy(a_d, a_h, m*n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, m*n * sizeof(float), hipMemcpyHostToDevice);
	// copy into surface memory
	transformGlobalToSurface<<<dimGrid, dimBlock>>>(a_d, aSurf, m, n); 
	transformGlobalToSurface<<<dimGrid, dimBlock>>>(b_d, bSurf, m, n); 
	hipEventRecord(finish, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time transfering to GPU = %.17f\n", elapsedTime);
	//time_transfering_to_gpu = elapsedTime;

	hipEventRecord(start, 0);
	for (int i=0; i<iter; i++){
		iterate_GPU_surface<<<dimGrid, dimBlock>>>(aSurf, bSurf, m, n);
		iterate_GPU_surface<<<dimGrid, dimBlock>>>(bSurf, aSurf, m, n);
	}
	hipEventRecord(finish, 0);
		
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time for compute on GPU = %.17f\n", elapsedTime);
	time_compute = elapsedTime;

	// copy into RAM
	hipEventRecord(start, 0);
	transformSurfaceToGlobal<<<dimGrid, dimBlock>>>(aSurf, a_d, m, n); 
	transformSurfaceToGlobal<<<dimGrid, dimBlock>>>(bSurf, b_d, m, n); 
	hipMemcpy(a_h, a_d, m*n * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(b_h, b_d, m*n * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(finish, 0);
	
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time to transfer to RAM = %.17f\n", elapsedTime);	
	//time_transfering_to_cpu = elapsedTime;

	float* thermometer_d;
	float* thermometer_h = (float*) malloc(m * sizeof(float));
	if (calc_avg_temp == 1){
		hipMalloc((void**)&thermometer_d, m * sizeof(float));
		hipMemset(thermometer_d, 0, m * sizeof(float));

		if (calc_avg_temp_no_atomic == 0){
			hipEventRecord(start, 0);
			calculate_avg_temp_GPU<<<dimGrid, dimBlock>>>(a_d, m, n, thermometer_d);
			hipEventRecord(finish, 0);
		} else { // from hw1
			hipEventRecord(start, 0);
 			sum_rows_gpu<<<dimGrid, dimBlock>>>(m, n, a_d, thermometer_d);
			hipEventRecord(finish, 0);

		}

		hipEventSynchronize(start);
		hipEventSynchronize(finish);
		hipEventElapsedTime(&elapsedTime, start, finish);
		printf("Time to calculate averages on GPU = %.17f\n", elapsedTime);	
		time_calc_averages = elapsedTime;

	}	
	hipMemcpy(thermometer_h, thermometer_d, m * sizeof(float), hipMemcpyDeviceToHost);

	

	// end //
	//=================================================================//
	
		
	clock_t time_start;			
	clock_t time_end;			
	float cpu_time_allocating;
	float cpu_time_compute;
	float cpu_time_calc_averages;

	float* a;
	float* b;
	float* thermometer;

	// CPU Calculation //
	//=================================================================//
	if (calc_cpu == 1){
			printf("\n//======================================//\n");
			printf("               CPU Calculation              \n");
			printf("//======================================//\n\n");

	
			// allocalte matrices a, b
			time_start = clock();
			a = (float*) malloc(m*n * sizeof(float));
			b = (float*) malloc(m*n * sizeof(float));
			time_end = clock();
			cpu_time_allocating = (float)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3);
			printf("Time allocating on CPU = %.17f\n", cpu_time_allocating);
			
			time_start = clock();
			init_matrix(a, m, n);
			init_matrix(b, m, n);
			time_end = clock();
			printf("Time initialising matrices on CPU = %.17f\n", (float)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3));
			
//			printf("before \n");
//			printf("printing a\n");
//			print_matrix(a, m, n);

			time_start = clock();
			for (int i=0; i<iter; i++){
				iterate(a, b, m, n);
				iterate(b, a, m, n);
			}
			time_end = clock();
			cpu_time_compute = (float)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3);
			printf("Time for compute on CPU = %.17f\n", cpu_time_compute);
				
			thermometer = (float*) calloc(m, sizeof(float));
			if (calc_avg_temp == 1){

				time_start = clock();
				calculate_avg_temp(a, m, n, thermometer);
				time_end = clock();
				cpu_time_calc_averages = (float)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3);
				printf("Time to calculate averages on CPU = %.17f\n", cpu_time_calc_averages);

			}	
	}
	// end //
	//=================================================================//

		

	if (show_timings_next_to_eachother == 1){
		// compute errors
		printf("\n");
		float max_matrix_diff = max_diff(a_h, a, m, n);
		float max_avg_diff = max_diff(thermometer_h, thermometer, m, 1);

		printf("\n//======================================//\n");
		printf("      SHOWING MAIN TIMINGS AND SPEEDUPS     \n");
		printf("//======================================//\n\n");
			
		printf("Allocating memory    | CPU: %f | GPU: %f | Speedup: %f\n", 
				cpu_time_allocating, time_allocating, cpu_time_allocating/time_allocating);
		printf("Main compute         | CPU: %f | GPU: %f | Speedup: %f\n", 
				cpu_time_compute, time_compute, cpu_time_compute/time_compute);
		printf("Calculating averages | CPU: %f | GPU: %f | Speedup: %f\n", 
				cpu_time_calc_averages, time_calc_averages, cpu_time_calc_averages/time_calc_averages);
		printf("\n");

		printf("Maximum difference between CPU and GPU | Avg temp: %f\n                                       | Matrices: %f\n",max_avg_diff, max_matrix_diff); 
		
		printf("\n");

		if (save_data == 1){
			FILE *fp = fopen(filename, "a");

			fprintf(fp, "%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f", 
					m, n, block_size_x, block_size_y,
					cpu_time_allocating, time_allocating, cpu_time_allocating/time_allocating,
					cpu_time_compute, time_compute, cpu_time_compute/time_compute,
					cpu_time_calc_averages, time_calc_averages, cpu_time_calc_averages/time_calc_averages);
			fprintf(fp, "\n");
			fclose(fp);
		}

//		printf("after \n");
//		printf("printing a_h\n");
//		print_matrix(a_h, m, n);
//		printf("printing a\n");
//		print_matrix(a, m, n);

	}

	// free cuda parts
	free(a_h);
	free(b_h);
	free(thermometer_h);
	hipFree(a_d);
	hipFree(b_d);
	hipFreeArray(array_a_d); // frees cuda surface
	hipFreeArray(array_b_d);
	hipFree(thermometer_d);

	// free cpu parts
	if (calc_cpu == 1){
		free(a);
		free(b);
		free(thermometer);
	}


    return 0;
}
