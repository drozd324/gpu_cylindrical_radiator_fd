#include "hip/hip_runtime.h"
#include "task2_funcs.h"

//__global__ void copySurface(hipSurfaceObject_t inputSurface, hipSurfaceObject_t outputSurface, int m, int n) {
//	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
//
//	if (idx<n && idy<m) {
//		double data;
//		surf2Dread(&data, inputSurface, idx*8, idy);
//		surf2Dwrite(data, outputSurface, idx*8, idy);
//	}
//}

__global__ void transformSurfaceToGlobal(hipSurfaceObject_t surface, double* gpu_data, int m, int n) {
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;

	if ((idx<m) && (idy<n)) {
		float temp = 0.0f;
		surf2Dread(&temp, surface, idx * 8, idy);  
		gpu_data[idx * n + idy] = (double) temp;

		//surf2Dread(&(gpu_data[idx*n + idy]), surface, idx*8 , idy);
    }
}

__global__ void transformGlobalToSurface(double* gpu_data, hipSurfaceObject_t surface, int m, int n) {
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;

	if ((idx<m) && (idy<n)) {
		surf2Dwrite((float)gpu_data[idx*n + idy], surface, idx*8, idy);
    }
}

__global__ void iterate_GPU_surface(hipSurfaceObject_t nextSurface, hipSurfaceObject_t prevSurface, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	float prevSurf_j_minus2; 
	float prevSurf_j_minus1;
	float prevSurf_j;
	float prevSurf_j_plus1;
	float prevSurf_j_plus2;

	double nextSurf_j;

	int mod_j_minus2;
	int mod_j_minus1;
	int mod_j_plus1;
	int mod_j_plus2;


	if ((idx<m) && (idy<n)){

		mod_j_minus2 = (idy - 2 + n) % n;
		mod_j_minus1 = (idy - 1 + n) % n;
		mod_j_plus1 = (idy + 1) % n;
		mod_j_plus2 = (idy + 2) % n;

		// read
		surf2Dread(&prevSurf_j_minus2, prevSurface, idx*8, mod_j_minus2);
		surf2Dread(&prevSurf_j_minus1, prevSurface, idx*8, mod_j_minus1);
		surf2Dread(&prevSurf_j       , prevSurface, idx*8, idy         );
		surf2Dread(&prevSurf_j_plus1 , prevSurface, idx*8, mod_j_plus1 );
		surf2Dread(&prevSurf_j_plus2 , prevSurface, idx*8, mod_j_plus2 );

		// compute
		nextSurf_j = ((1.60*(double)prevSurf_j_minus2) + (1.55*(double)prevSurf_j_minus1) + (double)prevSurf_j + (0.60*(double)prevSurf_j_plus1) + (0.25*(double)prevSurf_j_plus2)) / ((double)(5.0));
		
		// write
		surf2Dwrite((float)nextSurf_j, nextSurface, idx*8, idy);
	}
}

__global__ void calculate_avg_temp_GPU(double* matrix, int m, int n, double* thermometer){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if ((idx<m) && (idy<n)){
		atomicAdd(&(thermometer[idx]), matrix[idx*n + idy] / (double)n);
	}
}


// iteration function for global memory data
__global__ void iterate_GPU_global(double* nextMatrix, double* previousMatrix, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int mod_j_minus2;
	int mod_j_minus1;
	int mod_j_plus1;
	int mod_j_plus2;

	if ((idx<m) && (idy<n)){
		mod_j_minus2 = (idy - 2 + n) % n;
		mod_j_minus1 = (idy - 1 + n) % n;
		mod_j_plus1 = (idy + 1) % n;
		mod_j_plus2 = (idy + 2) % n;


		nextMatrix[idx*n + idy] = ((1.60*previousMatrix[idx*n + mod_j_minus2]) + 
									(1.55*previousMatrix[idx*n + mod_j_minus1]) + 
									previousMatrix[idx*n + idy] + 
									(0.60*previousMatrix[idx*n + mod_j_plus1]) +
									(0.25*previousMatrix[idx*n + mod_j_plus2]));
		nextMatrix[idy*n + idy] /= (double)(5.0);
	}
}


// hw1 reduce code

__global__ void sum_rows_gpu(int m, int n, double* a, double* v){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < m){
        v[idx] = 0;

       	for (int j=0; j<n; j++){ 
           	v[idx] += (0 < (a[idx*n + j])) ? a[idx*n + j] : -a[idx*n + j];
       	}
		
		v[idx] = v[idx] / n;     // added the divide in order for this function to copmute the average
	}
}

// hw1 reduce code end
