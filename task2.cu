#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include "task1_funcs.h"
#include "task2_funcs.h"

int main(int argc, char *argv[]) {

	int option;
	int m = 32;
	int n = 32;
	int iter = 10;
	int calc_cpu = 0;
	int calc_avg_temp = 0;
	int show_timings_next_to_eachother = 0;

	int block_size_x = 2; //threads per block
	int block_size_y = 2; //threads per block

    while ((option = getopt(argc, argv, "m:n:p:x:y:act")) != -1) {
        switch (option) {
            case 'm': // set num cols m of matrix
	            m = atoi(optarg);
				break;
			case 'n': // set num rows n of matrix
				n = atoi(optarg);
				break;
			case 'p': // set iterations
				iter = atoi(optarg);
				break;
			case 'x': // set block_size_x
				block_size_x = atoi(optarg);
				break;
			case 'y': // set block_size_y
				block_size_y = atoi(optarg);
				break;
			case 'a': // sets caclulation of average temperature for each row
				calc_avg_temp = 1;
				break;
			case 'c': // caclulates cpu version of algorithm
				calc_cpu = 1;
				break;
			case 't': // caclulates all timings
				show_timings_next_to_eachother = 1;
				calc_cpu = 1;
				calc_avg_temp = 1;
				break;
        }
    }

	
	// GPU Calculation //
	//=================================================================//
	printf("\n//======================================//\n");
	printf("              GPU Calculation               \n");
	printf("//======================================//\n\n");

	if ( ((n*m) % (block_size_x*block_size_y)) != 0){
		fprintf(stderr, "ERROR: block size (number of threads per block) doesnt divide the total size of the matrix\n");
		return 1;
	}

	int N = n;
	int M = m;
	dim3 dimBlock (block_size_x, block_size_y);
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1),(M/dimBlock.y) + (!(M%dimBlock.y)?0:1) );
		
	//cuda timings
	hipEvent_t start, finish;
	hipEventCreate(&start);
	hipEventCreate(&finish);
	float elapsedTime;

	float time_allocating;
	//float time_transfering_to_gpu;
	float time_compute;
	float time_calc_averages;
	//float time_transfering_to_cpu;

	// allocalte matrices a_h, b_h on host
	float* a_h;
	float* b_h;
	a_h = (float*) malloc(m*n * sizeof(float));
	b_h = (float*) malloc(m*n * sizeof(float));
	
	// alloc on device
	float* a_d;
	float* b_d;
	hipEventRecord(start, 0);
	hipMalloc((void**)&a_d, m*n * sizeof(float));
	hipMalloc((void**)&b_d, m*n * sizeof(float));
	hipEventRecord(finish, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time allocating on GPU = %.17f\n", elapsedTime);
	time_allocating = elapsedTime;

	// init on cpu
	init_matrix(a_h, m, n);
	init_matrix(b_h, m, n);

	// copy to gpu
	hipEventRecord(start, 0);
	hipMemcpy(a_d, a_h, m*n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b_h, m*n * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(finish, 0);

	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time transfering to GPU = %.17f\n", elapsedTime);
	//time_transfering_to_gpu = elapsedTime;

	hipEventRecord(start, 0);
	init_matrix_GPU<<<dimGrid, dimBlock>>>(a_d, m, n);
	init_matrix_GPU<<<dimGrid, dimBlock>>>(b_d, m, n);
	hipEventRecord(finish, 0);
	
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time initialising matrices on GPU = %.17f\n", elapsedTime);

	hipEventRecord(start, 0);
	for (int i=0; i<iter; i++){
		iterate_GPU<<<dimGrid, dimBlock>>>(a_d, b_d, m, n);
		iterate_GPU<<<dimGrid, dimBlock>>>(b_d, a_d, m, n);
	}
	hipEventRecord(finish, 0);
		
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time for compute on GPU = %.17f\n", elapsedTime);
	time_compute = elapsedTime;

	if (calc_avg_temp == 1){
		float* thermometer_d;
		hipMalloc((void**)&thermometer_d, m * sizeof(float));

		hipEventRecord(start, 0);
		calculate_avg_temp_GPU<<<dimGrid, dimBlock>>>(a_d, m, n, thermometer_d);
		hipEventRecord(finish, 0);

		hipEventSynchronize(start);
		hipEventSynchronize(finish);
		hipEventElapsedTime(&elapsedTime, start, finish);
		printf("Time to calculate averages on GPU = %.17f\n", elapsedTime);	
		time_calc_averages = elapsedTime;

		hipFree(thermometer_d);
	}	

	hipEventRecord(start, 0);
	hipMemcpy(a_h, a_d, m*n * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(b_h, b_d, m*n * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(finish, 0);
	
	hipEventSynchronize(start);
	hipEventSynchronize(finish);
	hipEventElapsedTime(&elapsedTime, start, finish);
	printf("Time to transfer to RAM = %.17f\n", elapsedTime);	
	//time_transfering_to_cpu = elapsedTime;
	
	// free
	free(a_h);	
	free(b_h);	
	hipFree(a_d);	
	hipFree(b_d);	

	// end //
	//=================================================================//
	
		
	clock_t time_start;			
	clock_t time_end;			
	float cpu_time_allocating;
	float cpu_time_compute;
	float cpu_time_calc_averages;

	// CPU Calculation //
	//=================================================================//
	if (calc_cpu == 1){
			printf("\n//======================================//\n");
			printf("               CPU Calculation              \n");
			printf("//======================================//\n\n");

	
			// allocalte matrices a, b
			float* a;
			float* b;
			time_start = clock();
			a = (float*) malloc(m*n * sizeof(float));
			b = (float*) malloc(m*n * sizeof(float));
			time_end = clock();
			cpu_time_allocating = (float)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3);
			printf("Time allocating on CPU = %.17f\n", cpu_time_allocating);
			
			time_start = clock();
			init_matrix(a, m, n);
			init_matrix(b, m, n);
			time_end = clock();
			printf("Time initialising matrices on CPU = %.17f\n", (float)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3));
			
			time_start = clock();
			for (int i=0; i<iter; i++){
				iterate(a, b, m, n);
				iterate(b, a, m, n);
			}
			time_end = clock();
			cpu_time_compute = (float)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3);
			printf("Time for compute on CPU = %.17f\n", cpu_time_compute);
				
			if (calc_avg_temp == 1){
				float* thermometer = (float*) calloc(m, sizeof(float));

				time_start = clock();
				calculate_avg_temp(a, m, n, thermometer);
				time_end = clock();
				cpu_time_calc_averages = (float)(time_end - time_start) / (CLOCKS_PER_SEC * 1e-3);
				printf("Time to calculate averages on CPU = %.17f\n", cpu_time_calc_averages);

				free(thermometer);
			}	
			
			// free matrices a, b
			free(a);
			free(b);
	}
	// end //
	//=================================================================//

	if (show_timings_next_to_eachother == 1){
		printf("\n//======================================//\n");
		printf("      SHOWING MAIN TIMINGS AND SPEEDUPS     \n");
		printf("//======================================//\n\n");
			
		printf("Allocating memory    | CPU: %.17f | GPU: %.17f | Speedup: %.17f\n", 
				cpu_time_allocating, time_allocating, cpu_time_allocating/time_allocating);
		printf("Main compute         | CPU: %.17f | GPU: %.17f | Speedup: %.17f\n", 
				cpu_time_compute, time_compute, cpu_time_compute/time_compute);
		printf("Calculating averages | CPU: %.17f | GPU: %.17f | Speedup: %.17f\n", 
				cpu_time_calc_averages, time_calc_averages, cpu_time_calc_averages/time_calc_averages);
		printf("\n");


		char filename[100];
		sprintf(filename, "writeup/task3.csv");
		FILE *fp = fopen(filename, "a");

		//fprintf(fp,"m,n,block_size_x,block_size_y,cpu_time_allocating,time_allocating,speedup_time_allocating,cpu_time_compute,time_compute,speedup_time_compute,cpu_time_calc_averages,time_calc_averages,speedup_time_calc_averages);

//				      1		2	3		4	5		6	7		8	9		10	  11	 12		13	
		fprintf(fp, "%d,%d,%d,%d,%.17f,%.17f,%.17f,%.17f,%.17f,%.17f,%.17f,%.17f,%.17f", 
				m, n, block_size_x, block_size_y,
				cpu_time_allocating, time_allocating, cpu_time_allocating/time_allocating,
				cpu_time_compute, time_compute, cpu_time_compute/time_compute,
				cpu_time_calc_averages, time_calc_averages, cpu_time_calc_averages/time_calc_averages);
		fprintf(fp, "\n");
		fclose(fp);

	}


    return 0;
}
