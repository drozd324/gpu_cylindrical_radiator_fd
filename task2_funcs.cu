#include "hip/hip_runtime.h"
#include "task2_funcs.h"

__global__ void init_matrix_GPU(float** matrix, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx<m){
		if (idy==0){
			matrix[idx][0] = 0.98 * (float)((i+1)*(i+1)) / (float)(n*n); 
		}
	}
	
	if (idx<m){
		if ((1<=idy) && (idy<n)){
			matrix[idx][idy] = matrix[idx][0] * ( ((float)((m-idy)*(m-idy))) / ((float)(m*m)));	
		}
	}
}

__global__ void iterate_GPU(float** nextMatrix, float** previousMatrix, int m, int n){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx<m){
		if ((2<=idy) && (idy<n-2)){
			nextMatrix[idx][idy] = ((1.60*previousMatrix[idx][idy-2]) + 
									(1.55*previousMatrix[idx][idy-1]) + 
									previousMatrix[idx][idy] + 
									(0.60*previousMatrix[idx][idy+1]) +
									(0.25*previousMatrix[idx][idy+2]));
			nextMatrix[idx][idy] /= (float)(5.0);
		}
	}
}

__global__ void calculate_avg_temp_GPU(float** matrix, int m, int n, float* thermometer){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx<m){
		float* loc_therm = calloc(n, sizeof(float));
		if (idy<n){
			atomicAdd(&(thermometer[idx]), matrix[idx][idy]);	
		}
		thermometer[i] /= m;
	}
}

